#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>
#include <unordered_map>
#include <cmath> // for abs

#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <string.h>
#include <algorithm>    // std::sort

// thrust
#include <thrust/host_vector.h>	
#include <thrust/device_vector.h>	
#include <thrust/generate.h>	
#include <thrust/sort.h>	
#include <thrust/copy.h>		
#include <thrust/inner_product.h>	
#include <thrust/gather.h>


#include <chrono> // timer
using namespace std;

unordered_map<string, int> hashmap;  // domain --> index
unordered_map<int, string> indices;  // index --> domain 
unordered_map<int, vector<int>> adj; // adjacency list for the graph
unordered_map<int, int> outdegree;
int idx = 0;
int nIterations = 0;
int occupiedRows = 0;

// CSR data structures
vector<int> row_begin;
vector<float> values;
vector<int> col_indices;
vector<int> rows;
vector<int> nonOccupiedRows;
// Constants
float alpha = 0.2;
float epsilon = 1e-3;  // not 1e-6 because of float precision

struct func
{
  __device__ float operator()(thrust::tuple<float, float> t) 
  {
     float f = thrust::get<0>(t) - thrust::get<1>(t);
     return abs(f);
  }
};


struct multiply_add
{
  __host__ __device__
  float operator()(float x) const
  {
    // define alpha as a constant
    const float alpha = 0.2f;

    // multiply x by alpha and add alpha - 1
    return x * alpha + (1.0f - alpha);
  }
};

// Reads the file and converts it to CSR format.
void toCSR(string file)
{
    ifstream stream(file);
    string line;

    while (getline(stream, line))
    {
        string from, to;
        stringstream ss(line);
        ss >> from;
        ss >> to;

        int f, t = 0;
        if (hashmap.find(from) == hashmap.end()){
            hashmap[from] = idx;
            indices[idx] = from;
            f = idx++;
        } else f = hashmap[from];
        if (hashmap.find(to) == hashmap.end()) {
            hashmap[to] = idx;
            indices[idx] = to;
            t = idx++;
        } else t = hashmap[to];

        if (adj.find(f) == adj.end()) adj[f] = {};
        if (adj.find(t) == adj.end()) adj[t] = {};

        adj[t].push_back(f); // we want Q_ij to be marked if there is a link from j(th) page to i(th) page.
        if(outdegree.find(f) == outdegree.end()) outdegree[f] = 1;
        else outdegree[f]++;
    }

    stream.close();

    row_begin.push_back(0);
    for (int i = 0; i < adj.size(); i++)
    {
        bool occupied = false;
        row_begin.push_back(row_begin.back() + adj[i].size());
        for (int j = 0; j < adj[i].size(); j++) {
            col_indices.push_back(adj[i][j]);
            rows.push_back(i);
            occupied = true;
        }
        if (occupied) occupiedRows++;
        else nonOccupiedRows.push_back(i);
    }

    int N = row_begin.size() - 1;
    for (int i = 0; i < N; i++)
    {
        for(int j = row_begin[i]; j < row_begin[i + 1]; j++) 
        {
            int from = col_indices[j];   // website T_i which has a citation to our website A
            float outd = outdegree[from];  // calculate C_i of the website (outdegree)
            values.push_back(1/outd);      // find in unordered_map could take worst case O(n), linked list structure faster or array 
        }
    }
}


// Returns the names of the first 5 hosts that have the highest rankings.
vector<string> getFive(thrust::device_vector<float>& r1) {
    thrust::device_vector<int> index(r1.size());
    thrust::sequence(index.begin(), index.end()); // 0, 1, 2, 3, ...
    thrust::sort_by_key(r1.begin(), r1.end(), index.begin());

    vector<string> firstFive;
    for(int i= index.size() - 1; i >= index.size() - 5; i--) {  
        int idx = index[i];   
        firstFive.push_back(indices[idx]);
    }
    return firstFive;
    
}

int main()
{   
    
    cout << "Input is read sequentially - it'll take a while." << endl;
    auto start = chrono::high_resolution_clock::now();
    toCSR("../graph.txt");
    auto stop = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::seconds>(stop - start);
    cout << "Total amount of time to read the file: " << duration.count() << " seconds" << endl;
    auto io = duration.count();

    std::ofstream myfile;
    myfile.open ("timings.csv");
    myfile << "I/O - CSR transformation, Pagerank Algorithm\n";

    
    start = chrono::high_resolution_clock::now();
    int N = row_begin.size() - 1;
    // thrust CSR
    nIterations = 0;
    thrust::device_vector<float> r1(N, 1.0);
    thrust::device_vector<float> col_indices_d(col_indices.begin(), col_indices.end());
    thrust::device_vector<float> values_d(values.begin(), values.end());
    thrust::device_vector<int> rows_d(rows.begin(), rows.end());
    thrust::device_vector<int> nonOccupied(nonOccupiedRows.begin(), nonOccupiedRows.end());
    while (true)
    {
        thrust::device_vector<float> r2(N, 0.0);
        thrust::device_vector<int> ind(occupiedRows, 0);
        thrust::device_vector<float> P(col_indices_d.size(), 0);

        // cout << col_indices.size() << endl;
        thrust::transform(
            thrust::make_permutation_iterator(r1.begin(), col_indices_d.begin()),
            thrust::make_permutation_iterator(r1.end(), col_indices_d.end()),
            values_d.begin(), P.begin(), thrust::multiplies<float>());

        thrust::reduce_by_key(
            rows_d.begin(), rows_d.end(),
            P.begin(), ind.begin(), r2.begin());
        // https://github.com/NVIDIA/thrust/issues/1621 ?
        thrust::device_vector<int> indextended = ind;
        indextended.reserve(nonOccupied.size());
        indextended.insert(indextended.end(), nonOccupied.begin(), nonOccupied.end());
        thrust::sort_by_key(indextended.begin(), indextended.end(), r2.begin());
        
        thrust::transform(r2.begin(), r2.end(), r2.begin(), multiply_add());
        float cSum = thrust::transform_reduce(
            thrust::make_zip_iterator(thrust::make_tuple(r1.begin(), r2.begin())), 
            thrust::make_zip_iterator(thrust::make_tuple(r1.end(), r2.end())), 
            func(), 0.0f, thrust::plus<float>());

        nIterations++;
        if (cSum <= epsilon) break;
        thrust::copy(r2.begin(), r2.end(), r1.begin());  // copy r2 to r1: r1 = r2
    }

    stop = chrono::high_resolution_clock::now();
    duration = chrono::duration_cast<chrono::seconds>(stop - start);
    myfile << io << "," << duration.count() << "\n";
    cout << "Total amount of time to calculate the pagerank: " << duration.count() << " seconds" << endl;

    vector<string> firstFive = getFive(r1);
    cout << "Names of the first 5 hosts that have the highest rankings (from first to fifth):" << endl;
    for(auto& c: firstFive) cout << c << "\n";
    
    myfile.close();
    return 0;
}

/**
 * 4mekp13kca78a3hfsrb0k813n9 0491md82hej8u15vi98isrmuih 3165mii1s1g0invqs94q303v0v 46o3c5beh6kiojkvr1tvsk4ptt 2494c7mt12frm3c3go86abe13h
*/